#include "niggli.cuh"
#include <ATen/cuda/HIPContext.h>
#include <chrono>
#include <pybind11/pybind11.h>
#include <torch/extension.h>


torch::Tensor niggliReduceTorch(torch::Tensor cell, float tol = 1e-5, int num_iterations = 1000)
{
    cell = cell.contiguous();
    hipSetDevice(cell.device().index());
    TORCH_CHECK(cell.device().is_cuda(), "Cell tensor must be on CUDA device");
    TORCH_CHECK(cell.size(1) == 3 && cell.size(2) == 3, "Cell tensor must have shape [N, 3, 3]");

    int num_cells = cell.size(0);

    torch::Tensor result = torch::empty({num_cells, 3, 3}, torch::dtype(torch::kFloat).device(cell.device()));

    hipDeviceSynchronize();

    niggliReduceKernel<<<1, num_cells>>>(
        reinterpret_cast<const float(*)[9]>(cell.data_ptr<float>()),
        reinterpret_cast<float(*)[9]>(result.data_ptr<float>()),
        tol,
        num_iterations);

    hipDeviceSynchronize();

    return result;
}

PYBIND11_MODULE(torch_niggli, m)
{
    m.def("niggli_reduce", &niggliReduceTorch, "A function that reduces cell using the Niggli reduction algorithm on the GPU.",
        pybind11::arg("cell"), pybind11::arg("tol") = 1e-5, pybind11::arg("num_iterations") = 1000);
}
