#include "hip/hip_runtime.h"
#include "niggli.cuh"
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__ __device__ double absDet(const double mat[9])
{
    double det = mat[0 * 3 + 0] * (mat[1 * 3 + 1] * mat[2 * 3 + 2] - mat[1 * 3 + 2] * mat[2 * 3 + 1]) - mat[0 * 3 + 1] * (mat[1 * 3 + 0] * mat[2 * 3 + 2] - mat[1 * 3 + 2] * mat[2 * 3 + 0]) + mat[0 * 3 + 2] * (mat[1 * 3 + 0] * mat[2 * 3 + 1] - mat[1 * 3 + 1] * mat[2 * 3 + 0]);
    return std::abs(det);
}

__device__ void multiplyAndUpdate(const double src[9], double dst[9])
{
    double temp[9];
    temp[0 * 3 + 0] = src[0 * 3 + 0] * dst[0 * 3 + 0] + src[0 * 3 + 1] * dst[1 * 3 + 0] + src[0 * 3 + 2] * dst[2 * 3 + 0];
    temp[0 * 3 + 1] = src[0 * 3 + 0] * dst[0 * 3 + 1] + src[0 * 3 + 1] * dst[1 * 3 + 1] + src[0 * 3 + 2] * dst[2 * 3 + 1];
    temp[0 * 3 + 2] = src[0 * 3 + 0] * dst[0 * 3 + 2] + src[0 * 3 + 1] * dst[1 * 3 + 2] + src[0 * 3 + 2] * dst[2 * 3 + 2];
    temp[1 * 3 + 0] = src[1 * 3 + 0] * dst[0 * 3 + 0] + src[1 * 3 + 1] * dst[1 * 3 + 0] + src[1 * 3 + 2] * dst[2 * 3 + 0];
    temp[1 * 3 + 1] = src[1 * 3 + 0] * dst[0 * 3 + 1] + src[1 * 3 + 1] * dst[1 * 3 + 1] + src[1 * 3 + 2] * dst[2 * 3 + 1];
    temp[1 * 3 + 2] = src[1 * 3 + 0] * dst[0 * 3 + 2] + src[1 * 3 + 1] * dst[1 * 3 + 2] + src[1 * 3 + 2] * dst[2 * 3 + 2];
    temp[2 * 3 + 0] = src[2 * 3 + 0] * dst[0 * 3 + 0] + src[2 * 3 + 1] * dst[1 * 3 + 0] + src[2 * 3 + 2] * dst[2 * 3 + 0];
    temp[2 * 3 + 1] = src[2 * 3 + 0] * dst[0 * 3 + 1] + src[2 * 3 + 1] * dst[1 * 3 + 1] + src[2 * 3 + 2] * dst[2 * 3 + 1];
    temp[2 * 3 + 2] = src[2 * 3 + 0] * dst[0 * 3 + 2] + src[2 * 3 + 1] * dst[1 * 3 + 2] + src[2 * 3 + 2] * dst[2 * 3 + 2];
    for (int i = 0; i < 9; ++i) {
        dst[i] = temp[i];
    }
}

__global__ void niggliReduceKernel(
    const double cell_src[][9],
    double cell_dst[][9],
    double tol,
    int num_iterations)
{
    long start = clock64();
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    double eps = tol * std::pow(absDet(cell_src[idx]), 1.0 / 3.0);
    double cell[9];
    for (int i = 0; i < 9; ++i) {
        cell[i] = cell_src[idx][i];
    }
    double M[9];
    double A, B, C, E, N, Y;
    int i;
    for (i = 0; i < num_iterations; ++i) {
        A = cell[0 * 3 + 0] * cell[0 * 3 + 0] + cell[0 * 3 + 1] * cell[0 * 3 + 1] + cell[0 * 3 + 2] * cell[0 * 3 + 2];
        B = cell[1 * 3 + 0] * cell[1 * 3 + 0] + cell[1 * 3 + 1] * cell[1 * 3 + 1] + cell[1 * 3 + 2] * cell[1 * 3 + 2];
        C = cell[2 * 3 + 0] * cell[2 * 3 + 0] + cell[2 * 3 + 1] * cell[2 * 3 + 1] + cell[2 * 3 + 2] * cell[2 * 3 + 2];
        E = 2 * (cell[1 * 3 + 0] * cell[2 * 3 + 0] + cell[1 * 3 + 1] * cell[2 * 3 + 1] + cell[1 * 3 + 2] * cell[2 * 3 + 2]);
        N = 2 * (cell[2 * 3 + 0] * cell[0 * 3 + 0] + cell[2 * 3 + 1] * cell[0 * 3 + 1] + cell[2 * 3 + 2] * cell[0 * 3 + 2]);
        Y = 2 * (cell[0 * 3 + 0] * cell[1 * 3 + 0] + cell[0 * 3 + 1] * cell[1 * 3 + 1] + cell[0 * 3 + 2] * cell[1 * 3 + 2]);

        // A1
        if (B + eps < A || std::abs(A - B) < eps && std::abs(E) > std::abs(N) + eps) {
            M[0 * 3 + 0] = 0, M[0 * 3 + 1] = -1, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = -1, M[1 * 3 + 1] = 0, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = 0, M[2 * 3 + 2] = -1;
            multiplyAndUpdate(M, cell);
        }

        A = cell[0 * 3 + 0] * cell[0 * 3 + 0] + cell[0 * 3 + 1] * cell[0 * 3 + 1] + cell[0 * 3 + 2] * cell[0 * 3 + 2];
        B = cell[1 * 3 + 0] * cell[1 * 3 + 0] + cell[1 * 3 + 1] * cell[1 * 3 + 1] + cell[1 * 3 + 2] * cell[1 * 3 + 2];
        C = cell[2 * 3 + 0] * cell[2 * 3 + 0] + cell[2 * 3 + 1] * cell[2 * 3 + 1] + cell[2 * 3 + 2] * cell[2 * 3 + 2];
        E = 2 * (cell[1 * 3 + 0] * cell[2 * 3 + 0] + cell[1 * 3 + 1] * cell[2 * 3 + 1] + cell[1 * 3 + 2] * cell[2 * 3 + 2]);
        N = 2 * (cell[2 * 3 + 0] * cell[0 * 3 + 0] + cell[2 * 3 + 1] * cell[0 * 3 + 1] + cell[2 * 3 + 2] * cell[0 * 3 + 2]);
        Y = 2 * (cell[0 * 3 + 0] * cell[1 * 3 + 0] + cell[0 * 3 + 1] * cell[1 * 3 + 1] + cell[0 * 3 + 2] * cell[1 * 3 + 2]);

        // A2
        if (C + eps < B || std::abs(B - C) < eps && std::abs(N) > std::abs(Y) + eps) {
            M[0 * 3 + 0] = -1, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = 0, M[1 * 3 + 2] = -1;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = -1, M[2 * 3 + 2] = 0;
            multiplyAndUpdate(M, cell);
            continue;
        }

        double ll = (std::abs(E) < eps) ? 0 : (E / std::abs(E));
        double m = (std::abs(N) < eps) ? 0 : (N / std::abs(N));
        double n = (std::abs(Y) < eps) ? 0 : (Y / std::abs(Y));

        if (ll * m * n == 1) {
            // A3
            double i = (ll == -1) ? -1 : 1;
            double j = (m == -1) ? -1 : 1;
            double k = (n == -1) ? -1 : 1;
            double M[9] = {i, 0.0, 0.0, 0.0, j, 0.0, 0.0, 0.0, k};
            M[0 * 3 + 0] = i, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = j, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = 0, M[2 * 3 + 2] = k;
            multiplyAndUpdate(M, cell);
        } else if (ll * m * n == 0 || ll * m * n == -1) {
            // A4
            double i = (ll == 1) ? -1 : 1;
            double j = (m == 1) ? -1 : 1;
            double k = (n == 1) ? -1 : 1;

            if (i * j * k == -1) {
                if (n == 0) {
                    k = -1;
                } else if (m == 0) {
                    j = -1;
                } else if (ll == 0) {
                    i = -1;
                }
            }
            M[0 * 3 + 0] = i, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = j, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = 0, M[2 * 3 + 2] = k;
            multiplyAndUpdate(M, cell);
        }

        A = cell[0 * 3 + 0] * cell[0 * 3 + 0] + cell[0 * 3 + 1] * cell[0 * 3 + 1] + cell[0 * 3 + 2] * cell[0 * 3 + 2];
        B = cell[1 * 3 + 0] * cell[1 * 3 + 0] + cell[1 * 3 + 1] * cell[1 * 3 + 1] + cell[1 * 3 + 2] * cell[1 * 3 + 2];
        C = cell[2 * 3 + 0] * cell[2 * 3 + 0] + cell[2 * 3 + 1] * cell[2 * 3 + 1] + cell[2 * 3 + 2] * cell[2 * 3 + 2];
        E = 2 * (cell[1 * 3 + 0] * cell[2 * 3 + 0] + cell[1 * 3 + 1] * cell[2 * 3 + 1] + cell[1 * 3 + 2] * cell[2 * 3 + 2]);
        N = 2 * (cell[2 * 3 + 0] * cell[0 * 3 + 0] + cell[2 * 3 + 1] * cell[0 * 3 + 1] + cell[2 * 3 + 2] * cell[0 * 3 + 2]);
        Y = 2 * (cell[0 * 3 + 0] * cell[1 * 3 + 0] + cell[0 * 3 + 1] * cell[1 * 3 + 1] + cell[0 * 3 + 2] * cell[1 * 3 + 2]);

        // A5
        if (std::abs(E) > B + eps || std::abs(E - B) < eps && 2 * N < Y - eps || std::abs(E + B) < eps && -eps > Y) {
            double s = E / std::abs(E);
            M[0 * 3 + 0] = 1, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = 1, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = -s, M[2 * 3 + 2] = 1;
            multiplyAndUpdate(M, cell);
            continue;
        }

        // A6
        if (std::abs(N) > A + eps || std::abs(N - A) < eps && 2 * E < Y - eps || std::abs(N + A) < eps && -eps > Y) {
            double s = N / std::abs(N);
            M[0 * 3 + 0] = 1, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = 1, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = -s, M[2 * 3 + 1] = 0, M[2 * 3 + 2] = 1;
            multiplyAndUpdate(M, cell);
            continue;
        }

        // A7
        if (std::abs(Y) > A + eps || std::abs(Y - A) < eps && 2 * E < N - eps || std::abs(Y + A) < eps && -eps > N) {
            double s = Y / std::abs(Y);
            M[0 * 3 + 0] = 1, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = -s, M[1 * 3 + 1] = 1, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 0, M[2 * 3 + 1] = 0, M[2 * 3 + 2] = 1;
            multiplyAndUpdate(M, cell);
            continue;
        }

        // A8
        if (-eps > E + N + Y + A + B || std::abs(E + N + Y + A + B) < eps && eps < Y + 2 * (A + N)) {
            M[0 * 3 + 0] = 1, M[0 * 3 + 1] = 0, M[0 * 3 + 2] = 0;
            M[1 * 3 + 0] = 0, M[1 * 3 + 1] = 1, M[1 * 3 + 2] = 0;
            M[2 * 3 + 0] = 1, M[2 * 3 + 1] = 1, M[2 * 3 + 2] = 1;
            multiplyAndUpdate(M, cell);
            continue;
        }
        break;
    }
    for (int i = 0; i < 9; ++i) {
        cell_dst[idx][i] = cell[i];
    }
}

std::vector<double> niggliReduce(const std::vector<double>& cell, double tol, int num_iterations)
{
    int num_elements = cell.size();
    int num_cells = num_elements / 9;

    double* cell_src = nullptr;
    double* cell_dst = nullptr;
    hipMalloc((void**)&cell_src, num_elements * sizeof(double));
    hipMalloc((void**)&cell_dst, num_elements * sizeof(double));

    hipMemcpy(cell_src, cell.data(), num_elements * sizeof(double), hipMemcpyHostToDevice);

    niggliReduceKernel<<<num_cells, 1>>>(reinterpret_cast<double(*)[9]>(cell_src), reinterpret_cast<double(*)[9]>(cell_dst), tol, num_iterations);
    hipDeviceSynchronize();

    std::vector<double> result(num_elements);
    hipMemcpy(result.data(), cell_dst, num_elements * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(cell_src);
    hipFree(cell_dst);

    return result;
}
